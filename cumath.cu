#include "hip/hip_runtime.h"
#include "cumath.h"

#define DEVICE_WRAPPER_ARG1(func)                       \
    __global__ void cuda_##func(double *x, double *y) { \
        *y = func(*x);                                  \
    }

#define HOST_WRAPPER_ARG1(func)                                      \
    double cu##func(double x) {                                      \
        double y;                                                    \
        double *d_x, *d_y;                                           \
        hipMalloc((void **) &d_x, sizeof(double));                  \
        hipMalloc((void **) &d_y, sizeof(double));                  \
        hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice); \
        cuda_##func<<<1,1>>>(d_x, d_y);                              \
        hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost); \
        hipFree(d_x);                                               \
        hipFree(d_y);                                               \
        return y;                                                    \
    }

#define DEVICE_WRAPPER_ARG2(func)                                  \
    __global__ void cuda_##func(double *x, double *y, double *z) { \
        *z = func(*x, *y);                                         \
    }

#define HOST_WRAPPER_ARG2(func)                                      \
    double cu##func(double x, double y) {                            \
        double z;                                                    \
        double *d_x, *d_y, *d_z;                                     \
        hipMalloc((void **) &d_x, sizeof(double));                  \
        hipMalloc((void **) &d_y, sizeof(double));                  \
        hipMalloc((void **) &d_z, sizeof(double));                  \
        hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice); \
        hipMemcpy(d_y, &y, sizeof(double), hipMemcpyHostToDevice); \
        cuda_##func<<<1,1>>>(d_x, d_y, d_z);                         \
        hipMemcpy(&z, d_z, sizeof(double), hipMemcpyDeviceToHost); \
        hipFree(d_x);                                               \
        hipFree(d_y);                                               \
        hipFree(d_z);                                               \
        return z;                                                    \
    }

/* constants */
const double cupi = 3.141592653589793238462643383279502884197;
const double cue  = 2.718281828459045235360287471352662497757;

/* power and logarithmic function */
DEVICE_WRAPPER_ARG1(exp);
HOST_WRAPPER_ARG1(exp);
DEVICE_WRAPPER_ARG1(expm1);
HOST_WRAPPER_ARG1(expm1);
__global__ void cuda_log(double *x, double *y, double *z) {*z = log(*x) / log(*y);}
HOST_WRAPPER_ARG2(log);
DEVICE_WRAPPER_ARG1(log1p);
HOST_WRAPPER_ARG1(log1p);
DEVICE_WRAPPER_ARG1(log2);
HOST_WRAPPER_ARG1(log2);
DEVICE_WRAPPER_ARG1(log10);
HOST_WRAPPER_ARG1(log10);
DEVICE_WRAPPER_ARG2(pow);
HOST_WRAPPER_ARG2(pow);
DEVICE_WRAPPER_ARG1(sqrt);
HOST_WRAPPER_ARG1(sqrt);

/* trigonometric function */
DEVICE_WRAPPER_ARG1(acos);
HOST_WRAPPER_ARG1(acos);
DEVICE_WRAPPER_ARG1(asin);
HOST_WRAPPER_ARG1(asin);
DEVICE_WRAPPER_ARG1(atan);
HOST_WRAPPER_ARG1(atan);
DEVICE_WRAPPER_ARG2(atan2);
HOST_WRAPPER_ARG2(atan2);
DEVICE_WRAPPER_ARG1(cos);
HOST_WRAPPER_ARG1(cos);
DEVICE_WRAPPER_ARG2(hypot);
HOST_WRAPPER_ARG2(hypot);
DEVICE_WRAPPER_ARG1(sin);
HOST_WRAPPER_ARG1(sin);
DEVICE_WRAPPER_ARG1(tan);
HOST_WRAPPER_ARG1(tan);

/* angular conversion */
__global__ void cuda_degrees(double *x, double *y) {*y = *x * 180 / cupi;}
HOST_WRAPPER_ARG1(degrees);
__global__ void cuda_radians(double *x, double *y) {*y = *x * cupi / 180;}
HOST_WRAPPER_ARG1(radians);

/* hyperbolic function */
DEVICE_WRAPPER_ARG1(acosh);
HOST_WRAPPER_ARG1(acosh);
DEVICE_WRAPPER_ARG1(asinh);
HOST_WRAPPER_ARG1(asinh);
DEVICE_WRAPPER_ARG1(atanh);
HOST_WRAPPER_ARG1(atanh);
DEVICE_WRAPPER_ARG1(cosh);
HOST_WRAPPER_ARG1(cosh);
DEVICE_WRAPPER_ARG1(sinh);
HOST_WRAPPER_ARG1(sinh);
DEVICE_WRAPPER_ARG1(tanh);
HOST_WRAPPER_ARG1(tanh);
