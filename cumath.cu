#include "hip/hip_runtime.h"
#include "cumath.h"

__global__ void cuda_sin(double *x, double *y) {
    *y = sin(*x);
}

__global__ void cuda_sinh(double *x, double *y) {
    *y = sinh(*x);
}

__global__ void cuda_sqrt(double *x, double *y) {
    *y = sqrt(*x);
}

__global__ void cuda_tan(double *x, double *y) {
    *y = tan(*x);
}

__global__ void cuda_tanh(double *x, double *y) {
    *y = tanh(*x);
}

double cusin(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_sin<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cusinh(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_sinh<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cusqrt(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_sqrt<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cutan(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_tanh<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cutanh(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_tanh<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

