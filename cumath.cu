#include "hip/hip_runtime.h"
#include "cumath.h"

const double pi = 3.141592653589793238462643383279502884197;

__global__ void cuda_degrees(double *x, double *y) {
    *y = *x * 180 / pi;
}

__global__ void cuda_radians(double *x, double *y) {
    *y = *x * pi / 180;  
}

__global__ void cuda_sin(double *x, double *y) {
    *y = sin(*x);
}

__global__ void cuda_sinh(double *x, double *y) {
    *y = sinh(*x);
}

__global__ void cuda_sqrt(double *x, double *y) {
    *y = sqrt(*x);
}

__global__ void cuda_tan(double *x, double *y) {
    *y = tan(*x);
}

__global__ void cuda_tanh(double *x, double *y) {
    *y = tanh(*x);
}

double cudegrees(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_degrees<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double curadians(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_radians<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cusin(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_sin<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cusinh(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_sinh<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cusqrt(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_sqrt<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cutan(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_tanh<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

double cutanh(double x) {
    double y;
    double *d_x, *d_y;

    hipMalloc((void **) &d_x, sizeof(double));
    hipMalloc((void **) &d_y, sizeof(double));

    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);

    cuda_tanh<<<1,1>>>(d_x, d_y);

    hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return y;
}

