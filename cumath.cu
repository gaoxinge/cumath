#include "hip/hip_runtime.h"
#include "cumath.h"

#define DEVICE_WRAPPER_ARG1(func)                       \
    __global__ void cuda_##func(double *x, double *y) { \
        *y = func(*x);                                  \
    }

#define HOST_WRAPPER_ARG1(func)                                      \
    double cu##func(double x) {                                      \
        double y;                                                    \
        double *d_x, *d_y;                                           \
        hipMalloc((void **) &d_x, sizeof(double));                  \
        hipMalloc((void **) &d_y, sizeof(double));                  \
        hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice); \
        cuda_##func<<<1,1>>>(d_x, d_y);                              \
        hipMemcpy(&y, d_y, sizeof(double), hipMemcpyDeviceToHost); \
        hipFree(d_x);                                               \
        hipFree(d_y);                                               \
        return y;                                                    \
    }

#define DEVICE_WRAPPER_ARG2(func)                                  \
    __global__ void cuda_##func(double *x, double *y, double *z) { \
        *z = func(*x, *y);                                         \
    }

#define HOST_WRAPPER_ARG2(func)                                      \
    double cu##func(double x, double y) {                            \
        double z;                                                    \
        double *d_x, *d_y, *d_z;                                     \
        hipMalloc((void **) &d_x, sizeof(double));                  \
        hipMalloc((void **) &d_y, sizeof(double));                  \
        hipMalloc((void **) &d_z, sizeof(double));                  \
        hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice); \
        hipMemcpy(d_y, &y, sizeof(double), hipMemcpyHostToDevice); \
        cuda_##func<<<1,1>>>(d_x, d_y, d_z);                         \
        hipMemcpy(&z, d_z, sizeof(double), hipMemcpyDeviceToHost); \
        hipFree(d_x);                                               \
        hipFree(d_y);                                               \
        hipFree(d_z);                                               \
        return z;                                                    \
    }

/* constants */
const double cupi = 3.141592653589793238462643383279502884197;

/* power and logarithmic function */
DEVICE_WRAPPER_ARG1(sqrt);
HOST_WRAPPER_ARG1(sqrt);

/* trigonometric function */
DEVICE_WRAPPER_ARG1(acos);
HOST_WRAPPER_ARG1(acos);
DEVICE_WRAPPER_ARG1(asin);
HOST_WRAPPER_ARG1(asin);
DEVICE_WRAPPER_ARG1(atan);
HOST_WRAPPER_ARG1(atan);
DEVICE_WRAPPER_ARG2(atan2);
HOST_WRAPPER_ARG2(atan2);
DEVICE_WRAPPER_ARG1(cos);
HOST_WRAPPER_ARG1(cos);
DEVICE_WRAPPER_ARG2(hypot);
HOST_WRAPPER_ARG2(hypot);
DEVICE_WRAPPER_ARG1(sin);
HOST_WRAPPER_ARG1(sin);
DEVICE_WRAPPER_ARG1(tan);
HOST_WRAPPER_ARG1(tan);

/* angular conversion */
__global__ void cuda_degrees(double *x, double *y) {*y = *x * 180 / cupi;}
HOST_WRAPPER_ARG1(degrees);
__global__ void cuda_radians(double *x, double *y) {*y = *x * cupi / 180;}
HOST_WRAPPER_ARG1(radians);

/* hyperbolic function */
DEVICE_WRAPPER_ARG1(acosh);
HOST_WRAPPER_ARG1(acosh);
DEVICE_WRAPPER_ARG1(asinh);
HOST_WRAPPER_ARG1(asinh);
DEVICE_WRAPPER_ARG1(atanh);
HOST_WRAPPER_ARG1(atanh);
DEVICE_WRAPPER_ARG1(cosh);
HOST_WRAPPER_ARG1(cosh);
DEVICE_WRAPPER_ARG1(sinh);
HOST_WRAPPER_ARG1(sinh);
DEVICE_WRAPPER_ARG1(tanh);
HOST_WRAPPER_ARG1(tanh);
